#include "hip/hip_runtime.h"
#include <iostream> 
#include <cmath>
#include <chrono>
#include <vector>
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>
#include <boost/program_options.hpp>
#include <nvtx3/nvToolsExt.h>

namespace po = boost::program_options;

#define BLOCK_THREADS 512
#define CHECK_INTERVAL 1000

void initialize_boundaries(double* x, int N) {
    x[0] = 10.0;
    x[N - 1] = 20.0;
    x[N * N - 1] = 30.0;
    x[N * (N - 1)] = 20.0;

    for (int j = 0; j < N; ++j) {
        x[j] = 10.0 + (20.0 - 10.0) * j / (N - 1);
        x[(N - 1) * N + j] = 30.0 + (20.0 - 30.0) * j / (N - 1);
    }

    for (int i = 0; i < N; ++i) {
        x[i * N] = 10.0 + (20.0 - 10.0) * i / (N - 1);
        x[i * N + (N - 1)] = 20.0 + (30.0 - 20.0) * i / (N - 1);
    }
}

__global__ void update_kernel(double* d_x, double* d_x_new, double* d_diff, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N-1 && j < N-1) {
        int idx = i * N + j;
        d_x_new[idx] = 0.25 * (d_x[(i - 1) * N + j] + d_x[(i + 1) * N + j] +
                                d_x[i * N + (j - 1)] + d_x[i * N + (j + 1)]);
        d_diff[idx] = fabs(d_x_new[idx] - d_x[idx]);
    }
}

__global__ void copy_kernel(double* d_x, double* d_x_new, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N-1 && j < N-1) {
        int idx = i * N + j;
        d_x[idx] = d_x_new[idx];
    }
}

__global__ void block_max_kernel(double* d_input, int total, double* d_output, int num_blocks) {
    typedef hipcub::BlockReduce<double, BLOCK_THREADS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    BlockReduce reduce_inst(temp_storage);
    int idx = threadIdx.x + blockIdx.x * BLOCK_THREADS;
    double val = (idx < total) ? d_input[idx] : -__builtin_inf();
    double block_max = reduce_inst.Reduce(val, hipcub::Max());
    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = block_max;
    }
}


void solve(int N, double epsilon, int max_iter) {
    double* x = new double[N * N]();
    initialize_boundaries(x, N);

    
    hipStream_t stream;
    hipStreamCreate(&stream);

    double *d_x, *d_x_new, *d_diff;
    hipMallocAsync((void**)&d_x, N * N * sizeof(double), stream); 
    hipMallocAsync((void**)&d_x_new, N * N * sizeof(double), stream);
    hipMallocAsync((void**)&d_diff, (N-2) * (N-2) * sizeof(double),stream);
    hipMemcpy(d_x, x, N * N * sizeof(double), hipMemcpyHostToDevice);

    int total = (N-2)*(N-2);
    std::vector<int> temp_sizes;
    int current_size = total;
    while (current_size > 1) {
        int num_blocks = (current_size + BLOCK_THREADS - 1) / BLOCK_THREADS;
        temp_sizes.push_back(num_blocks);
        current_size = num_blocks;
    }

    double* d_temp_buffer;
    size_t temp_buffer_size = 0;
    for (auto s : temp_sizes) temp_buffer_size += s;
    hipMallocAsync(&d_temp_buffer, temp_buffer_size * sizeof(double), stream);

    std::vector<double*> temp_arrays(temp_sizes.size());
    size_t offset = 0;
    for (size_t i = 0; i < temp_sizes.size(); ++i) {
        temp_arrays[i] = d_temp_buffer + offset;
        offset += temp_sizes[i];
    }

    dim3 block_update(16, 16);
    dim3 grid_update((N-2 + block_update.x - 1) / block_update.x, (N-2 + block_update.y - 1) / block_update.y);

    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    nvtxRangePushA("update_kernel");
    update_kernel<<<grid_update, block_update, 0, stream>>>(d_x, d_x_new, d_diff, N);
    nvtxRangePop();

    double* current_array = d_diff;
    current_size = total;

    nvtxRangePushA("block_max_kernel_reduction");
    for (size_t i = 0; i < temp_sizes.size(); ++i) {
        int num_blocks = temp_sizes[i];
        block_max_kernel<<<num_blocks, BLOCK_THREADS, 0, stream>>>(current_array, current_size, temp_arrays[i], num_blocks);
        current_array = temp_arrays[i];
        current_size = num_blocks;
    }
    nvtxRangePop();

    nvtxRangePushA("copy_kernel");
    copy_kernel<<<grid_update, block_update, 0, stream>>>(d_x, d_x_new, N);
    hipStreamEndCapture(stream, &graph);
    nvtxRangePop();

    hipGraphExec_t instance;
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    int iter = 0;
    double residual = std::numeric_limits<double>::max();
    auto start = std::chrono::high_resolution_clock::now();

    nvtxRangePushA("solving");
    double max_error = 0.0;

    while (residual > epsilon && iter < max_iter) {
        for (int k = 0; k < CHECK_INTERVAL && iter < max_iter; ++k) {
            hipGraphLaunch(instance, stream);
            iter++;
        }

        hipMemcpyAsync(&max_error, temp_arrays.back(), sizeof(double), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);  
        residual = max_error;
    }
    nvtxRangePop();

    auto end = std::chrono::high_resolution_clock::now();

    hipMemcpy(x, d_x, N * N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_x_new);
    hipFree(d_diff);
    hipFree(d_temp_buffer);
    hipGraphExecDestroy(instance);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Итераций: " << iter << ", Остаточная ошибка: " << residual
              << ", Время: " << elapsed.count() << " с\n";

    if (N == 10 || N == 13) {
        std::cout << "Итоговая сетка:\n";
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << x[i * N + j] << "\t";
            }
            std::cout << "\n";
        }
    }

    delete[] x;
}

int main(int argc, char* argv[]) {
    nvtxRangePushA("Init context");
    hipSetDevice(3);
    nvtxRangePop();

    int N;
    double epsilon;
    int max_iter;

    try {
        po::options_description desc("Допустимые опции");
        desc.add_options()
            ("help,h", "показать справку")
            ("size,N", po::value<int>(&N)->default_value(128), "размер сетки (N)")
            ("epsilon,e", po::value<double>(&epsilon)->default_value(1e-6), "эпсилон")
            ("max_iter,m", po::value<int>(&max_iter)->default_value(1000000), "максимум итераций");

        po::variables_map vm;
        po::store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);

        if (vm.count("help")) {
            std::cout << desc << "\n";
            return 1;
        }
        solve(N, epsilon, max_iter);

    } catch (std::exception& e) {
        std::cerr << "Ошибка: " << e.what() << "\n";
        return 1;
    }

    return 0;
}